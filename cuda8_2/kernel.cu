#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
	hipError_t cudaStatus;
	int N = 1 << 20;
	float *x, *y, *d_x, *d_y;
	//first, allocate memory in RAM
	x = (float*)malloc(N * sizeof(float));
	y = (float*)malloc(N * sizeof(float));

	//second, allocate memory in GPU buffer
	cudaStatus = hipMalloc(&d_x, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for d_x failed with error: %d", cudaStatus);
		goto Error;
	}
	cudaStatus = hipMalloc(&d_y, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for d_y failed with error: %d", cudaStatus);
		goto Error;
	}
	//initialize array in RAM
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	//move arrays from RAM to GPU buffer
	cudaStatus = hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy for transferring x to d_x failed with error: %d", cudaStatus);
		goto Error;
	}
	cudaStatus = hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy for transferring y to d_y failed with error: %d", cudaStatus);
		goto Error;
	}

	// Perform SAXPY on 1M elements in GPU
	saxpy << <(N + 255) / 256, 256 >> >(N, 2.0f, d_x, d_y);

	// Copy output vector from GPU buffer to host memory.
	hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
	{
		maxError = max(maxError, abs(y[i] - 4.0f));
		//printf("%f\t", y[i]);
	}

	printf("Max error: %f\n", maxError);

	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);

Error:
	hipFree(d_x);
	hipFree(d_y);
	//free(x);
	//free(y);
	return cudaStatus;
}
